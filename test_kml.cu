#include <xtensor/xarray.hpp>

#include <tiny-cuda-nn/common.h>
#include <tiny-cuda-nn/config.h>

#include <nlohmann/json.hpp>

// #include "src/defs.h"

#include "src/common.h"
#include "src/data_generator.h"


using json = nlohmann::json;

using namespace tcnn;


const uint32_t batch_size = 512;  // 1 << 16;
const uint32_t n_epochs = 150;

const uint32_t n_input_dims_to_encode = 9;  // (timesteps, cols, timeframes)
const uint32_t n_input_dims_to_pass_through = 2;  // TODO 2?
//const uint32_t n_input_dims = 3;  // TODO pass through dims?
const uint32_t n_row_size = 127458;  // TODO ? 127458=73*291*6

const uint32_t n_output_dims = 1;


bool generate_training_batch(DataGenerator& data_gen, GPUMatrix<float>& inputs, GPUMatrix<float>& outputs)
{
	xt::xarray<float> a_inputs;
	xt::xarray<float> a_outputs;

	if (!data_gen.next(a_inputs, a_outputs))
		return false;  // no more data

	//std::cout << "Inputs shape: " << a_inputs.shape() << std::endl;
	//std::cout << "Outputs shape: " << a_outputs.shape() << std::endl;
	
	//std::cout << "sizeof(a_inputs): " << a_inputs.size() * sizeof(float) << std::endl;
	//std::cout << "sizeof(a_outputs): " << a_outputs.size() * sizeof(float) << std::endl;
	//std::cout << "sizeof(inputs): " << inputs.n_bytes() << std::endl;
	//std::cout << "sizeof(outputs): " << outputs.n_bytes() << std::endl;
	
	CUDA_CHECK_THROW(hipMemcpy(inputs.data(), a_inputs.data(), a_inputs.size() * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_THROW(hipMemcpy(outputs.data(), a_outputs.data(), a_outputs.size() * sizeof(float), hipMemcpyHostToDevice));

	//std::cout << "Copied..." << std::endl;

	return true;
}

bool generate_pred_inputs(DataGenerator& data_gen, GPUMatrix<float>& inputs)
{
	// data_gen.reset();  // TODO fix reset

	xt::xarray<float> a_inputs;
	xt::xarray<float> a_outputs;
	
	if (!data_gen.next(a_inputs, a_outputs))
		return false;  // no more data

	std::cout << "Prediction inputs shape: " << a_inputs.shape() << std::endl;

	//std::cout << "sizeof(a_inputs): " << a_inputs.size() * sizeof(float) << std::endl;
	//std::cout << "sizeof(inputs): " << inputs.n_bytes() << std::endl;

	CUDA_CHECK_THROW(hipMemcpy(inputs.data(), a_inputs.data(), a_inputs.size() * sizeof(float), hipMemcpyHostToDevice));

	std::cout << "Prediction copied..." << std::endl;

	return true;
}


int main(int argc, char* argv[])
{
	json config = {
		{"loss", {
			{"otype", "L2"}
		}},
		{"optimizer", {
			{"otype", "Adam"},
			{"learning_rate", 1e-3},
		}},
		{"encoding", {
			{"otype", "OneBlob"},
			{"n_bins", 32},
		}},
		{"network", {
			{"otype", "FullyFusedMLP"},
			{"n_neurons", 128},
			{"n_hidden_layers", 5},
			{"activation", "ReLU"},
			{"output_activation", "Sigmoid"},
		}},
	};

	auto tpl = create_from_config(n_input_dims_to_encode, n_input_dims_to_pass_through, n_output_dims, config);
	auto losses = std::get<0>(tpl);
	auto optimizer = std::get<1>(tpl);
	auto network = std::get<2>(tpl);
	auto trainer = std::get<3>(tpl);

	// Prepare data generator
	
	// Train the model
	GPUMatrix<float> training_batch_inputs(n_row_size, batch_size);  // TODO n_input_dims?
	GPUMatrix<float> training_batch_targets(n_output_dims, batch_size);  // TODO n_output_dims?

	training_batch_inputs.initialize_constant(0);
	training_batch_targets.initialize_constant(0);
	/*hipMemset(&training_batch_inputs, 0, sizeof training_batch_inputs);
	hipMemset(&training_batch_targets, 0, sizeof training_batch_targets);*/

	//DataGenerator data_gen(batch_size, "pyfiles", "TRAIN.*.merged.npy");
	//generate_training_batch(data_gen, training_batch_inputs, training_batch_targets);
	//return 0;

	// create a cuda stream
	hipStream_t stream1;
	hipStreamCreate(&stream1);

	int epoch = 0;
	for (int i = 0; i < n_epochs; ++i) {
		std::cout << "load file..." << std::endl;
		DataGenerator data_gen(batch_size, "pyfiles", "TRAIN.*.merged.npy");
		std::cout << "loaded file." << std::endl;
		while (generate_training_batch(data_gen, training_batch_inputs, training_batch_targets))
		{
			float loss;
			trainer->training_step(stream1, training_batch_inputs, training_batch_targets, &loss);
			std::cout << "epoch=" << epoch << " iteration=" << i << " loss=" << loss << std::endl;
		}
		epoch++;
	}

	// wait until all cuda operations are complete
	hipStreamSynchronize(stream1);

	DataGenerator data_gen_eval(batch_size, "pyfiles", "EVAL.*.merged.npy");

	// Use the model
	GPUMatrix<float> inference_inputs(n_row_size, batch_size);
	training_batch_inputs.initialize_constant(0);
	generate_pred_inputs(data_gen_eval, inference_inputs);
	
	GPUMatrix<float> inference_outputs(n_output_dims, batch_size);
	network->inference(stream1, inference_inputs, inference_outputs);

	// wait until all cuda operations are complete
	hipStreamSynchronize(stream1);

	//std::cout << "inference_outputs[0]: " << *inference_outputs.data() << std::endl;

}